#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<Windows.h>

#include <GL/glew.h> // THIS MUST BE ABOVE gl.h
#include<GL/gl.h>

#include<hip/hip_runtime.h>
#include<cuda_gl_interop.h>
// CUDA utilities and system includes


#include "OGL.h"
#include "Sphere.h"
#include "vmath.h"
using namespace vmath;

// OpenGL Libraries
#pragma comment(lib, "glew32.lib")
#pragma comment(lib, "OpenGL32.lib")
#pragma comment(lib,"Sphere.lib")
#pragma comment(lib, "cudart.lib")

#define WIN_WIDTH 800
#define WIN_HEIGHT 600
#define FBO_WIDTH 512
#define FBO_HEIGHT 512

// Global Function Declarations
LRESULT CALLBACK WndProc(HWND, UINT, WPARAM, LPARAM);

// Global Variable declarations
HWND ghwnd = NULL;
HDC ghdc = NULL;
HGLRC ghrc = NULL;

BOOL gbActiveWindow = FALSE;
BOOL gbFullScreen = FALSE;
FILE *gpFile = NULL;

// Programable pipeline related global variables
GLuint shaderProgramObj;
int winWidth;
int winHeight;


enum 
{
    MVD_ATTRIBUTE_POSITION = 0,
    MVD_ATTRIBUTE_COLOR,
    MVD_ATTRIBUTE_NORMAL,
    MVD_ATTRIBUTE_TEXTURE0
};

GLuint vao_cube;
GLuint vbo_cube_position;
GLuint vbo_cube_texcoord;
GLuint mvpMatrixUniform;
GLuint texture_checkerboard;
GLuint textureSamplerUniform;

mat4 perspectiveProjectionMatrix;

GLfloat angleCube = 0.0f;

// FBO Related variables
GLuint fbo;
GLuint rbo;
GLuint fbo_texture;
bool bfboResult = false;

GLuint vbo_gpu;
hipError_t cudaResult;
struct hipGraphicsResource *graphicResource = NULL;
BOOL onGPU = FALSE;

unsigned int size_tex_data;
unsigned int num_texels;
unsigned int num_values;

// for proper depth test while rendering the scene
GLuint tex_screen;      // where we render the image
GLuint tex_cudaResult;  // where we will copy the CUDA result

float rotate[3];

char *ref_file = NULL;
bool enable_cuda = true;
bool animate = true;
int blur_radius = 3/2;
int max_blur_radius = 16;

unsigned int *cuda_dest_resource;
GLuint shDrawTex;  // draws a texture
struct hipGraphicsResource *cuda_tex_result_resource;
extern hipTextureObject_t inTexObject;
struct hipGraphicsResource *cuda_tex_screen_resource;

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                                   hipArray *g_data, unsigned int *g_odata,
                                   int imgw, int imgh, int tilew, int radius,
                                   float threshold, float highlight);

//texture Scene global variables 

// Programable pipeline related global variables
GLuint shaderProgram_sphere;

GLuint vao_sphere;
GLuint vbo_position_sphere;
GLuint vbo_normal_sphere;
GLuint vbo_elements_sphere;

GLuint modelMatrixUniform__sphere;
GLuint viewMatrixUniform__sphere;
GLuint projectionMatrixUniform__sphere;

GLuint laUniform_sphere[3];
GLuint ldUniform_sphere[3];
GLuint lsUniform_sphere[3];
GLuint lightPositionUniform_sphere[3];

GLuint kaUniform_sphere;
GLuint kdUniform_sphere;
GLuint ksUniform_sphere;
GLuint materiaShininessUniform_sphere;

GLuint lightingEnabledUniform_sphere;

BOOL bLight = FALSE;

float sphere_vertices[1146];
float sphere_normals[1146];
float sphere_textures[764];
unsigned short sphere_elements[2280];

unsigned int numVertices_sphere;
unsigned int numElements_sphere;

struct Light
{
    vec4 lightAmbient;
    vec4 lightDiffused;
    vec4 lightSpecular;
    vec4 lightPosition;
};

Light lights[3]; // Two different lights

mat4 perspectiveProjectionMatrix_sphere;

GLfloat materialAmbient_sphere[] = {0.0f, 0.0f, 0.0f, 0.0f};
GLfloat materialDiffused_sphere[] = {1.0f, 1.0f, 1.0f, 1.0f};
GLfloat materialSpecular_sphere[] = {1.0f, 1.0f, 1.0f, 1.0f};
GLfloat materialShininess_sphere = 128.0f;

GLfloat lightAngleZero_sphere = 0.0f;
GLfloat lightAngleOne_sphere = 0.0f;
GLfloat lightAngleTwo_sphere = 0.0f;
bool enable_cuda_postProcess = false;

int kernel[9];


GLubyte cpuConvolutionArray[FBO_WIDTH][FBO_HEIGHT][4];


GLfloat remap(GLfloat x, GLfloat in_min, GLfloat in_max, GLfloat out_min, GLfloat out_max) {
	return (x - in_min) * (out_max - out_min) / (in_max - in_min) + out_min;
}

float clampToCPUFloat(float x, float a, float b) { return max(a, min(b, x)); }

int clampToCPUInt(int x, int a, int b) { return max(a, min(b, x)); }

int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpszCmdLine, int iCmdShow)
{
    // Function Declarations
    int initialize(void);
    void display(void);
    void update(void);
    void uninitialize(void);

    // Varible Declarations
    HWND hwnd;
    MSG msg;
    TCHAR szAppName[] = TEXT("My Window");
    WNDCLASSEX wndClass;
    BOOL bDone = FALSE;
    int iRetVal = 0;

    int iScreenX = GetSystemMetrics(SM_CXSCREEN);
    int iScreenY =  GetSystemMetrics(SM_CYSCREEN);
    
    //code
    if (fopen_s(&gpFile, "Log.txt", "w") != 0)
    {
        MessageBox(NULL, TEXT("Log File Creation Failed... Exiting Now!!!"), TEXT("I/O Error"), MB_OK);
        exit(0);
    }
    else
    {
        fprintf(gpFile, "Log File is Created Succesfully\n");
    }

    wndClass.cbSize = sizeof(WNDCLASSEX);
    wndClass.cbClsExtra = 0;
    wndClass.cbWndExtra = 0;
    wndClass.hInstance = hInstance;
    wndClass.lpfnWndProc = WndProc;
    wndClass.lpszMenuName = NULL;
    wndClass.lpszClassName = szAppName;
    wndClass.hbrBackground = (HBRUSH) GetStockObject(BLACK_BRUSH);
    wndClass.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(MYICON));
    wndClass.hCursor = LoadCursor(NULL, IDC_ARROW);
    wndClass.style = CS_HREDRAW | CS_VREDRAW | CS_OWNDC;
    wndClass.hIconSm = LoadIcon(hInstance, MAKEINTRESOURCE(MYICON));
    
    // Register Window
    RegisterClassEx(&wndClass);

    hwnd =  CreateWindowEx(WS_EX_APPWINDOW,
        szAppName, 
        TEXT("MVD : OGL Window!"), 
        WS_OVERLAPPEDWINDOW | WS_CLIPCHILDREN | WS_CLIPSIBLINGS | WS_VISIBLE, 
        (iScreenX/2) - (WIN_WIDTH/2),
        (iScreenY/2) - (WIN_HEIGHT/2),
        WIN_WIDTH,
        WIN_HEIGHT,
        NULL,
        NULL,
        hInstance,
        NULL);

    ghwnd = hwnd;

    // Initialize
    iRetVal = initialize();

    if (iRetVal == -1)
    {
        fprintf(gpFile, "Choose Pixel Format Failed!\n");
        uninitialize();
    }
    else if (iRetVal == -2)
    {
        fprintf(gpFile, "Set Pixel Format Failed!\n");
        uninitialize();
    }
    else if (iRetVal == -3)
    {
        fprintf(gpFile, "Create OpenGL Context Failed!\n");
        uninitialize();
    }
    else if (iRetVal == -4)
    {
        fprintf(gpFile, "Making OpenGL Context as Current Context Failed!\n");
        uninitialize();
    }
    else if (iRetVal == -5)
    {
        fprintf(gpFile, "Glew Init() Failed!\n");
        uninitialize();
    }
    else if (iRetVal == -6)
    {
        fprintf(gpFile, "Texture Loading Failed!\n");
        uninitialize();
    }

    ShowWindow(hwnd, iCmdShow);

    // Forgrounding and Focusing the window
    SetForegroundWindow(hwnd); // Both ghwnd and hwnd will work here, but since hwnd is local here that's why we're using the same
    SetFocus(hwnd);

    while (bDone == FALSE)
    {
        if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
        {
            if (msg.message == WM_QUIT)
            {
                bDone = TRUE;
            }
            else
            {
                TranslateMessage(&msg);
                DispatchMessage(&msg);
            }
        }
        else
        {
            if (gbActiveWindow)
            {
                // Render the scene
                display();

                // Update the scene
                update();
            }
        }
    }

    // Janmejay and Takshak. Indray Swah, Takshkay swah
    uninitialize();
    return ((int) msg.wParam);
}

LRESULT CALLBACK WndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam)
{
    // Function Declarations
    void ToggleFullScreen(void);
    void resize(int, int);

    // Code
    
    switch (iMsg)
    {
        case WM_CHAR:
            switch (wParam)
            {
                case 'F':
                case 'f':
                    ToggleFullScreen();
                break;
                case 'L':
                case 'l':
                    if (bLight == FALSE)
                    {
                        bLight = TRUE;
                    }
                    else
                    {
                        bLight = FALSE;
                    }
                break;
                case '+':
                if (blur_radius < 16) {
                    blur_radius++;
                }
                break;
                case '-':
                if (blur_radius > 1) {
                    blur_radius--;
                }                
                break;
                case ' ':
                    enable_cuda_postProcess = !enable_cuda_postProcess;
                break;
                default:
                break;
            }
        break;
        case WM_KEYDOWN:
            switch (wParam)
            {
                case 27:
                    DestroyWindow(hwnd);
                break;
            
                default:
                break;
            }
        break;
        case WM_SETFOCUS:
            gbActiveWindow = TRUE;
        break;
        case WM_KILLFOCUS:
            gbActiveWindow = FALSE;
        break;
        case WM_ERASEBKGND:
            fprintf(gpFile, "ERASE BKGND is Called. \n");
            return 0;
        case WM_SIZE:
            // Every message has it's own unique information which is passed using LPARAM 
            // Here LOWORD : Width of the Window, HIWORD: Height of the Window
            resize(LOWORD(lParam), HIWORD(lParam));
        break;
        case WM_CLOSE:
            DestroyWindow(hwnd);
        break;
        case WM_DESTROY:
            PostQuitMessage(0);
        break;
    
        default:
        break;
    }
    return (DefWindowProc(hwnd, iMsg, wParam, lParam));
}

void ToggleFullScreen(void)
{
    // Variable Declarations
    static DWORD dwStyle;
    static WINDOWPLACEMENT wp;
    MONITORINFO mi;

    // Code
    fprintf(gpFile, "Entering ToggleFullScreen().\n");
    wp.length = sizeof(WINDOWPLACEMENT);

    if (gbFullScreen == FALSE)
    {
        dwStyle = GetWindowLong(ghwnd, GWL_STYLE);
        if (dwStyle & WS_OVERLAPPEDWINDOW)
        {
            mi.cbSize = sizeof(MONITORINFO);
            if (GetWindowPlacement(ghwnd, &wp) && 
                GetMonitorInfo(MonitorFromWindow(ghwnd, MONITORINFOF_PRIMARY), &mi))
            {
                SetWindowLong(ghwnd, GWL_STYLE, dwStyle & ~WS_OVERLAPPEDWINDOW);
                SetWindowPos(ghwnd, 
                    HWND_TOP, 
                    mi.rcMonitor.left, 
                    mi. rcMonitor.top, 
                    mi.rcMonitor.right - mi.rcMonitor.left, 
                    mi.rcMonitor.bottom - mi.rcMonitor.top, SWP_NOZORDER | SWP_FRAMECHANGED);
            }
        }
        ShowCursor(FALSE);
        gbFullScreen = TRUE;
    }
    else
    {
        SetWindowLong(ghwnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
        SetWindowPlacement(ghwnd, &wp);
        SetWindowPos(ghwnd, HWND_TOP, 0, 0, 0, 0, SWP_NOMOVE | SWP_NOOWNERZORDER | SWP_NOZORDER | SWP_NOSIZE | SWP_FRAMECHANGED);
        ShowCursor(TRUE);
        gbFullScreen = FALSE;
    }
}

////////////////////////////////////////////////////////////////////////////////
void createTextureDst(GLuint *tex_cudaResult, unsigned int size_x,
                      unsigned int size_y) {
  // create a texture
  glGenTextures(1, tex_cudaResult);
  glBindTexture(GL_TEXTURE_2D, *tex_cudaResult);

  // set basic parameters
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);


  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, size_x, size_y, 0,
               GL_RGB, GL_UNSIGNED_BYTE, NULL);
  // register this texture with CUDA
  hipGraphicsGLRegisterImage(
      &cuda_tex_result_resource, *tex_cudaResult, GL_TEXTURE_2D,
      cudaGraphicsMapFlagsWriteDiscard);
}



// copy image and process using CUDA
void processImage()
{
    void process(int, int, int);

  // run the Cuda kernel
  process(FBO_WIDTH, FBO_HEIGHT, blur_radius);

// CUDA generated data in cuda memory or in a mapped PBO made of BGRA 8 bits
// 2 solutions, here :
// - use glTexSubImage2D(), there is the potential to loose performance in
// possible hidden conversion
// - map the texture and blit the result thanks to CUDA API

  // We want to copy cuda_dest_resource data to the texture
  // map buffer objects to get CUDA device pointers
  hipArray *texture_ptr;
  hipGraphicsMapResources(1, &cuda_tex_result_resource, 0);
  hipGraphicsSubResourceGetMappedArray(
      &texture_ptr, cuda_tex_result_resource, 0, 0);

  int num_texels = FBO_WIDTH * FBO_HEIGHT;
  int num_values = num_texels * 4;
  int size_tex_data = sizeof(GLubyte) * num_values;
  hipMemcpyToArray(texture_ptr, 0, 0, cuda_dest_resource,
                                    size_tex_data, hipMemcpyDeviceToDevice);

  hipGraphicsUnmapResources(1, &cuda_tex_result_resource, 0);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void process(int width, int height, int radius) {
  hipArray *in_array;
  unsigned int *out_data;
  out_data = cuda_dest_resource;

  // map buffer objects to get CUDA device pointers
  hipGraphicsMapResources(1, &cuda_tex_screen_resource, 0);
  // printf("Mapping tex_in\n");
  hipGraphicsSubResourceGetMappedArray(
      &in_array, cuda_tex_screen_resource, 0, 0);

  // calculate grid size
  dim3 block(16, 16, 1);
  // dim3 block(16, 16, 1);
  dim3 grid(width / block.x, height / block.y, 1);
  int sbytes = (block.x + (2 * radius)) * (block.y + (2 * radius)) *
               sizeof(unsigned int);

  // execute CUDA kernel
  launch_cudaProcess(grid, block, sbytes, in_array, out_data, width, height,
                     block.x + (2 * radius), radius, 0.8f, 4.0f);

  hipGraphicsUnmapResources(1, &cuda_tex_screen_resource, 0);
  hipDestroyTextureObject(inTexObject);
}

void initCUDABuffers(int imgWidth, int imgHeight)
{
  // set up vertex data parameter
  num_texels = imgWidth * imgHeight;
  num_values = num_texels * 4;
  size_tex_data = sizeof(GLubyte) * num_values;
  hipMalloc((void **)&cuda_dest_resource, size_tex_data);
}

int initialize(void)
{
    // Function declarations
    void resize(int, int);
    void printGLInfo(void);
    void uninitialize(void);
    bool createFBO(GLint, GLint);
    int initialize_sphere(int, int);
    void initCUDABuffers(int, int);
    void genCPUTexture(void);

    // Variable declarations
    PIXELFORMATDESCRIPTOR pfd;
    int iPixelFormatIndex = 0;

    // Code
    ZeroMemory(&pfd, sizeof(PIXELFORMATDESCRIPTOR));
 
    // Initialization of PIXELFORMATDESCRIPTOR
    pfd.nSize = sizeof(PIXELFORMATDESCRIPTOR);
    pfd.nVersion = 1;
    pfd.dwFlags = PFD_DRAW_TO_WINDOW | PFD_SUPPORT_OPENGL | PFD_DOUBLEBUFFER;
    pfd.iPixelType = PFD_TYPE_RGBA;
    pfd.cColorBits = 32;
    pfd.cRedBits = 8;
    pfd.cGreenBits = 8;
    pfd.cBlueBits = 8;
    pfd.cAlphaBits = 8;
    pfd.cDepthBits = 32;

    // Get DC
    ghdc = GetDC(ghwnd);

    // Choose Pixel Format
    iPixelFormatIndex = ChoosePixelFormat(ghdc, &pfd);

    // if above call is successfull then it returns positive value
    if (iPixelFormatIndex == 0)
    {
        return(-1);
    }
    
    // Set the chosen pixel format
    if (SetPixelFormat(ghdc, iPixelFormatIndex, &pfd) == FALSE)
    {
        return(-2) ;
    }
    
    // Create OpenGL rendering context
    // Divyatwacade Janare pahile pahul
    ghrc = wglCreateContext(ghdc);
        
    if (ghrc == NULL)
    {
        return(-3);
    }

    // Make Rendering Context as Current Context
    // This is bridging API as hdc is not aware of OpenGL
    if (wglMakeCurrent(ghdc, ghrc) == FALSE)
    {
        return(-4);
    }

    // glew initalization
    if (glewInit() != GLEW_OK)
    {
        return (-5);
    }
    
    // Print OpenGL Info
    printGLInfo();


    int dev_count = 0;
    // CUDA Init
    cudaResult = hipGetDeviceCount(&dev_count);

    if (cudaResult != hipSuccess)
    {
        fprintf(gpFile, "CUDA hipGetDeviceCount() failed");
        uninitialize();
        exit(EXIT_FAILURE);
    }
    else if (dev_count == 0)
    {
        fprintf(gpFile, "No CUDA supported devices\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }
        
    // Select CUDA supported Device
    hipSetDevice(0); // Selecting the default 0th CUDA supported device

    // Vertex Shader
    const GLchar* vertexShaderSrcCode = 
        "#version 460 core" \
        "\n" \
        "in vec4 a_position;" \
        "in vec2 a_texcoord;" \
        "\n" \
        "uniform mat4 u_mvpMatrix;" \
        "out vec2 a_texcoord_out;" \
        "\n" \
        "void main(void)" \
        "\n" \
        "{" \
            "gl_Position = u_mvpMatrix * a_position;" \
            "a_texcoord_out = a_texcoord;" \
            "\n" \
        "}";

    // Create the Vertex Shader object.
    GLuint vertexShaderObj = glCreateShader(GL_VERTEX_SHADER);

    // Give the shader source to shader object.
    // Actually 3rd parameter is array if you have multiple shader source code
    // However, we have only one source code string
    glShaderSource(vertexShaderObj, 1, (const GLchar **)&vertexShaderSrcCode, NULL);

    // Compile the Shader source code for GPU format
    glCompileShader(vertexShaderObj);

    GLint status;
    GLint infoLogLength;
    char* log = NULL;

    glGetShaderiv(vertexShaderObj, GL_COMPILE_STATUS, &status);

    // If there is an error
    if (status == GL_FALSE)
    {
        glGetShaderiv(vertexShaderObj, GL_INFO_LOG_LENGTH, &infoLogLength);
        if (infoLogLength > 0)
        {
            log = (char*) malloc(infoLogLength);
            if (log != NULL)
            {
                GLsizei written;
                glGetShaderInfoLog(vertexShaderObj, infoLogLength, &written, log);
                fprintf(gpFile, "Vertex Shader Compilation Log: %s\n", log);
                free(log);
                log = NULL;
                uninitialize();
            }
        }
    }

    // Fragement Shader
    const GLchar* fragmentShaderSrcCode = 
        "#version 460 core" \
        "\n" \
        "in vec2 a_texcoord_out;" \
        "uniform sampler2D u_textureSampler;" \
        "out vec4 FragColor;" \
        "\n" \
        "vec4 color;"
        "\n" \
        "void main(void)" \
        "{" \
            "color = texture(u_textureSampler, a_texcoord_out);\n" \
            "FragColor = color ;" \
            "\n" \
        "}";
    
     // Create the Fragment Shader object.
    GLuint fragementShaderObj = glCreateShader(GL_FRAGMENT_SHADER);

    // Give the shader source to shader object.
    // Actually 3rd parameter is array if you have multiple shader source code
    // However, we have only one source code string
    glShaderSource(fragementShaderObj, 1, (const GLchar **)&fragmentShaderSrcCode, NULL);

    // Compile the Shader source code for GPU format
    glCompileShader(fragementShaderObj);

    status = 0;
    infoLogLength = 0;
    log = NULL;

    glGetShaderiv(fragementShaderObj, GL_COMPILE_STATUS, &status);

    // If there is an error
    if (status == GL_FALSE)
    {
        glGetShaderiv(fragementShaderObj, GL_INFO_LOG_LENGTH, &infoLogLength);
        if (infoLogLength > 0)
        {
            log = (char*) malloc(infoLogLength);
            if (log != NULL)
            {
                GLsizei written;
                glGetShaderInfoLog(fragementShaderObj, infoLogLength, &written, log);
                fprintf(gpFile, "Fragment Shader Compilation Log: %s\n", log);
                free(log);
                log = NULL;
                uninitialize();
            }
        }
    }

    // Shader Program Object
    shaderProgramObj = glCreateProgram();
    
    // Attach desired shader object to the program object
    glAttachShader(shaderProgramObj, vertexShaderObj);
    glAttachShader(shaderProgramObj, fragementShaderObj);

    // Pre-linked binding of Shader program object
    glBindAttribLocation(shaderProgramObj, MVD_ATTRIBUTE_POSITION, "a_position");
    glBindAttribLocation(shaderProgramObj, MVD_ATTRIBUTE_TEXTURE0, "a_texcoord");

    // Link the program
    glLinkProgram(shaderProgramObj);

    status = 0;
    infoLogLength = 0;
    log = NULL;

    glGetProgramiv(shaderProgramObj, GL_LINK_STATUS, &status);

    if (status == GL_FALSE)
    {
        glGetProgramiv(shaderProgramObj, GL_INFO_LOG_LENGTH, &infoLogLength);

        if (infoLogLength > 0)
        {
            log = (char*) malloc(infoLogLength);

            if (log != NULL)
            {
                GLsizei written;

                glGetProgramInfoLog(shaderProgramObj, infoLogLength, &written, log);
                fprintf(gpFile, "Shader Program Link Log: %s\n", log);
                free(log);
                uninitialize();
            }
        }
    }

    // Why post linking
    // Because without shaders get attached to shader program object it will not know
    mvpMatrixUniform = glGetUniformLocation(shaderProgramObj, "u_mvpMatrix");
    textureSamplerUniform = glGetUniformLocation(shaderProgramObj, "u_textureSampler");


    const GLfloat cubePosition[] =
    {
        // top
        1.0f, 1.0f, -1.0f,
        -1.0f, 1.0f, -1.0f, 
        -1.0f, 1.0f, 1.0f,
        1.0f, 1.0f, 1.0f,  

        // bottom
        1.0f, -1.0f, -1.0f,
       -1.0f, -1.0f, -1.0f,
       -1.0f, -1.0f,  1.0f,
        1.0f, -1.0f,  1.0f,

        // front
        1.0f, 1.0f, 1.0f,
       -1.0f, 1.0f, 1.0f,
       -1.0f, -1.0f, 1.0f,
        1.0f, -1.0f, 1.0f,

        // back
        1.0f, 1.0f, -1.0f,
       -1.0f, 1.0f, -1.0f,
       -1.0f, -1.0f, -1.0f,
        1.0f, -1.0f, -1.0f,

        // right
        1.0f, 1.0f, -1.0f,
        1.0f, 1.0f, 1.0f,
        1.0f, -1.0f, 1.0f,
        1.0f, -1.0f, -1.0f,

        // left
        -1.0f, 1.0f, 1.0f,
        -1.0f, 1.0f, -1.0f, 
        -1.0f, -1.0f, -1.0f, 
        -1.0f, -1.0f, 1.0f
    };

    const GLfloat cubeTexcoords[] = 
    {
        0.0f, 0.0f,
        1.0f, 0.0f,
        1.0f, 1.0f,
        0.0f, 1.0f,

        0.0f, 0.0f,
        1.0f, 0.0f,
        1.0f, 1.0f,
        0.0f, 1.0f,

        0.0f, 0.0f,
        1.0f, 0.0f,
        1.0f, 1.0f,
        0.0f, 1.0f,

        0.0f, 0.0f,
        1.0f, 0.0f,
        1.0f, 1.0f,
        0.0f, 1.0f,

        0.0f, 0.0f,
        1.0f, 0.0f,
        1.0f, 1.0f,
        0.0f, 1.0f,

        0.0f, 0.0f,
        1.0f, 0.0f,
        1.0f, 1.0f,
        0.0f, 1.0f,

    };

    // vao_cube
    glGenVertexArrays(1, &vao_cube);
    glBindVertexArray(vao_cube);

    //vbo_cube_position related code
    glGenBuffers(1, &vbo_cube_position);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_cube_position);

    glBufferData(GL_ARRAY_BUFFER, sizeof(cubePosition), cubePosition, GL_STATIC_DRAW);
    glVertexAttribPointer(MVD_ATTRIBUTE_POSITION, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(MVD_ATTRIBUTE_POSITION);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // vbo_square_color related color
    glGenBuffers(1, &vbo_cube_texcoord);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_cube_texcoord);
    glBufferData(GL_ARRAY_BUFFER, sizeof(cubeTexcoords), cubeTexcoords, GL_STATIC_DRAW);
    glVertexAttribPointer(MVD_ATTRIBUTE_TEXTURE0, 2, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(MVD_ATTRIBUTE_TEXTURE0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glBindVertexArray(0);

    // Create CUDA-OpenGL interoperability resource
    // Get my OpenGL Buffer as your Graphics Resource, make it writable and discard it after the use
    //cudaResult = hipGraphicsGLRegisterBuffer(&graphicResource, vbo_gpu, cudaGraphicsMapFlagsWriteDiscard);
    initCUDABuffers(FBO_WIDTH, FBO_HEIGHT);
    // if (cudaResult != hipSuccess)
    // {
    //     fprintf(gpFile, "CUDA hipGraphicsGLRegisterBuffer() failed!\n");
    //     uninitialize();
    //     exit(EXIT_FAILURE);
    // }

    // Required Depth and clear color related changes
    glClearDepth(1.0f);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL);

    // In Programable pipeline below 2 lines are deprecated
    //glShadeModel(GL_SMOOTH);
    //glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);

    // Here Starts OpenGL code
    // this doesn't actually Clear, but actually tells that which Color (blue in  this case) 
    // to be used when we do actual clear
    glClearColor(1.0f, 1.0f, 1.0f, 1.0f);

    glEnable(GL_TEXTURE_2D);

    perspectiveProjectionMatrix = mat4::identity();

    resize(WIN_WIDTH, WIN_HEIGHT);

    //cpuConvolutionArray = (unsigned int*) malloc(FBO_WIDTH * FBO_HEIGHT * 4 * sizeof(unsigned int));
    // FBO Code
    int iRetval;
    createTextureDst(&tex_cudaResult, FBO_WIDTH, FBO_HEIGHT);

    bfboResult = createFBO(FBO_WIDTH, FBO_HEIGHT);
    if (bfboResult == true)
    {
        iRetval = initialize_sphere(FBO_WIDTH, FBO_HEIGHT);
        if (iRetval)
        {
            fprintf(gpFile, "initialize_sphere Failed!!");
            return (-6);
        }        
    }
    else
    {
        fprintf(gpFile, "Create FBO Failed!!");
        return (-6);
    }
    
    kernel[0] = 1;
    kernel[1] = 2;
    kernel[2] = 1;
    kernel[3] = 2;
    kernel[4] = 4;
    kernel[5] = 2;
    kernel[6] = 1;
    kernel[7] = 2;
    kernel[8] = 1;

    genCPUTexture();
    return(0);
}

void printGLInfo()
{
    // Variable Declarations
    GLint numExtensions = 0;

    // Code
    fprintf(gpFile, "OpenGL Vendor: %s\n", glGetString(GL_VENDOR));
    fprintf(gpFile, "OpenGL Renderer: %s\n", glGetString(GL_RENDERER));
    fprintf(gpFile, "OpenGL Version: %s\n", glGetString(GL_VERSION));
    fprintf(gpFile, "GLSL Version: %s\n", glGetString(GL_SHADING_LANGUAGE_VERSION));
    glGetIntegerv(GL_NUM_EXTENSIONS, &numExtensions);

    fprintf(gpFile, "Number of Supported Extensions: %d\n", numExtensions);
    for (int i = 0; i < numExtensions; i++)
    {
        fprintf(gpFile, "%s\n", glGetStringi(GL_EXTENSIONS, i));
    }
}

bool createFBO(GLint textureWidth, GLint textureHeight)
{
    // Code
    void uninitialize(void);
    //1. Check available render buffer Size
    int maxRenderbufferSize;

    glGetIntegerv(GL_MAX_RENDERBUFFER_SIZE, &maxRenderbufferSize);
    if (maxRenderbufferSize < textureWidth || maxRenderbufferSize < textureHeight)
    {
        fprintf(gpFile, "Insufficient Render buffer size");
        return false;
    }
    
    //2. Create frame buffer object
    glGenFramebuffersEXT(1, &fbo);
    glBindFramebufferEXT(GL_FRAMEBUFFER_EXT, fbo);

    // 3. Create Render Buffer object
    glGenRenderbuffersEXT(1, &rbo);
    glBindRenderbufferEXT(GL_RENDERBUFFER_EXT, rbo);

    // 4. Storage and Format of the Render Buffer
    //This has nothing to with depth. 
    glRenderbufferStorage(GL_RENDERBUFFER_EXT, GL_DEPTH_COMPONENT24, textureWidth, textureHeight);

    //5. Create Empty texture for upcoming target scene
    glGenTextures(1, &fbo_texture);
    glBindTexture(GL_TEXTURE_2D, fbo_texture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA16F, textureWidth, textureHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glFramebufferTexture2DEXT(GL_FRAMEBUFFER_EXT, GL_COLOR_ATTACHMENT0_EXT, GL_TEXTURE_2D, fbo_texture, 0);
  
    // 6. Give RBO to FBO
    glFramebufferRenderbufferEXT(GL_FRAMEBUFFER_EXT,GL_DEPTH_ATTACHMENT_EXT, GL_RENDERBUFFER_EXT, rbo);

    // 7. Check whether FB created successfully or not
    GLenum result = glCheckFramebufferStatus(GL_FRAMEBUFFER_EXT);
    if (result != GL_FRAMEBUFFER_COMPLETE)
    {
        fprintf(gpFile, "Framebuffer is not complete \n");
        return false;
    }
    
    glBindFramebuffer(GL_FRAMEBUFFER_EXT, 0);
    cudaResult = hipGraphicsGLRegisterImage(&cuda_tex_screen_resource,
                                              fbo_texture, GL_TEXTURE_2D,
                                              cudaGraphicsMapFlagsReadOnly);
    if (cudaResult != hipSuccess)
    {
        fprintf(gpFile, "CUDA hipGraphicsGLRegisterImage() failed!\n");
        uninitialize();
        exit(EXIT_FAILURE);
    }
    return true;
}

int initialize_sphere(int width, int height)
{
    // Function declarations
    void resize_sphere(int, int);
    void recordAndBindBuffers(void);
    void getUniformsForShaderProgramForFragment(void);

#pragma region Per Fragment Shaders
     // Vertex Shader
    const GLchar* vertexShader_pfSrcCode = 
       "#version 460 core" \
        "\n" \
        "in vec4 a_position;" \
        "\n" \
        "in vec3 a_normal;" \
        "\n" \
        "uniform mat4 u_modelMatrix;" \
        "\n" \
        "uniform mat4 u_viewMatrix;" \
        "\n" \
        "uniform mat4 u_projectionMatrix;" \
        "\n" \
        "uniform vec4 u_lightPosition[3];" \
        "\n" \
        "uniform int u_lightingEnabled;" \
        "\n" \
        "out vec3 transformedNormals;" \
        "\n" \
        "out vec3 viewerVector;" \
        "\n" \
        "out vec3 lightDirection[3];\n" \
        "\n" \
        "void main(void)" \
        "\n" \
        "{\n" \
            "if(u_lightingEnabled == 1)\n" \
            "{\n" \
                // Goraud
                "vec4 eyeCordinates = u_viewMatrix * u_modelMatrix * a_position;\n" \
                "mat3 normalMatrix = mat3((u_viewMatrix * u_modelMatrix));\n" \
                "transformedNormals = normalMatrix * a_normal;\n" \
                "viewerVector = (-eyeCordinates.xyz);\n" \

                "for(int i = 0; i < 3; i++)" \
                "{\n" \
                    "lightDirection[i] = vec3(u_lightPosition[i]) - eyeCordinates.xyz;\n" \
                "}\n" \
            "}\n" \
            
            "gl_Position = u_projectionMatrix * u_viewMatrix * u_modelMatrix * a_position;" \
            "\n" \
        "}\n";

    // Create the Vertex Shader object.
    GLuint vertexShader_pfObj = glCreateShader(GL_VERTEX_SHADER);

    // Give the shader source to shader object.
    // Actually 3rd parameter is array if you have multiple shader source code
    // However, we have only one source code string
    glShaderSource(vertexShader_pfObj, 1, (const GLchar **)&vertexShader_pfSrcCode, NULL);

    // Compile the Shader source code for GPU format
    glCompileShader(vertexShader_pfObj);

    int status = 0;
    int infoLogLength = 0;
    char* log = NULL;

    glGetShaderiv(vertexShader_pfObj, GL_COMPILE_STATUS, &status);

    // If there is an error
    if (status == GL_FALSE)
    {
        glGetShaderiv(vertexShader_pfObj, GL_INFO_LOG_LENGTH, &infoLogLength);
        if (infoLogLength > 0)
        {
            log = (char*) malloc(infoLogLength);
            if (log != NULL)
            {
                GLsizei written;
                glGetShaderInfoLog(vertexShader_pfObj, infoLogLength, &written, log);
                fprintf(gpFile, "Vertex Shader Compilation Log: %s\n", log);
                free(log);
                log = NULL;
            }
        }
    }

    // Fragement Shader
    const GLchar* fragmentShader_pfSrcCode = 
       "#version 460 core" \
        "\n" \
        "in vec3 transformedNormals;" \
        "\n" \
        "in vec3 viewerVector;" \
        "\n" \
        "in vec3 lightDirection[3];\n" \
        "uniform vec3 u_la[3];" \
        "\n" \
        "uniform vec3 u_ld[3];" \
        "\n" \
        "uniform vec3 u_ls[3];" \
        "\n" \
        "uniform vec3 u_ka;" \
        "\n" \
        "uniform vec3 u_kd;" \
        "\n" \
        "uniform vec3 u_ks;" \
        "\n" \
        "uniform float u_materialShininess;" \
        "\n" \
        "uniform int u_lightingEnabled;" \
        "\n" \
        "vec3 phong_ads_light;" \
        "\n" \
        "out vec4 FragColor;" \
        "\n" \
        "void main(void)\n" \
        "{\n" \
            "vec3 ambient[3];\n" \
            "vec3 diffused[3];\n" \
            "vec3 reflectionVector[3];\n" \
            "vec3 specular[3]; \n" \
            "vec3 normalized_lightDirection[3];\n" \
            "vec3 normalized_transformed_normals = normalize(transformedNormals);\n" \
            "vec3 normalized_viewerVector = normalize(viewerVector);\n" \
            "if(u_lightingEnabled == 1)\n" \
            "{\n" \
                "for(int i = 0; i < 3; i++)" \
                "{\n" \
                    "normalized_lightDirection[i] = normalize(lightDirection[i]);\n" \
                    "ambient[i] = u_la[i] * u_ka;\n" \
                    "diffused[i] = u_ld[i] * u_kd * max(dot(normalized_lightDirection[i], normalized_transformed_normals), 0.0);\n" \
                    "reflectionVector[i] = reflect(-normalized_lightDirection[i], normalized_transformed_normals);\n" \
                    "specular[i] = u_ls[i] * u_ks * pow(max(dot(reflectionVector[i], normalized_viewerVector), 0.0), u_materialShininess);\n" \

                    "phong_ads_light += ambient[i] + diffused[i] + specular[i];\n" \
                "}\n" \
            "}\n" \
            "else\n" \
            "{" \
                "phong_ads_light = vec3(1.0, 1.0, 1.0);\n" \
            "}\n" \

            "FragColor = vec4(phong_ads_light, 1.0);" \
            "\n" \
        "}\n";
    
     // Create the Fragment Shader object.
    GLuint fragementShader_pfObj = glCreateShader(GL_FRAGMENT_SHADER);

    // Give the shader source to shader object.
    // Actually 3rd parameter is array if you have multiple shader source code
    // However, we have only one source code string
    glShaderSource(fragementShader_pfObj, 1, (const GLchar **)&fragmentShader_pfSrcCode, NULL);

    // Compile the Shader source code for GPU format
    glCompileShader(fragementShader_pfObj);

    status = 0;
    infoLogLength = 0;
    log = NULL;

    glGetShaderiv(fragementShader_pfObj, GL_COMPILE_STATUS, &status);

    // If there is an error
    if (status == GL_FALSE)
    {
        glGetShaderiv(fragementShader_pfObj, GL_INFO_LOG_LENGTH, &infoLogLength);
        if (infoLogLength > 0)
        {
            log = (char*) malloc(infoLogLength);
            if (log != NULL)
            {
                GLsizei written;
                glGetShaderInfoLog(fragementShader_pfObj, infoLogLength, &written, log);
                fprintf(gpFile, "Sphere Fragment Shader Compilation Log: %s\n", log);
                free(log);
                log = NULL;
            }
        }
    }

    // Shader Program Object
    shaderProgram_sphere = glCreateProgram();
    
    // Attach desired shader object to the program object
    glAttachShader(shaderProgram_sphere, vertexShader_pfObj);
    glAttachShader(shaderProgram_sphere, fragementShader_pfObj);

    // Pre-linked binding of Shader program object
    glBindAttribLocation(shaderProgram_sphere, MVD_ATTRIBUTE_POSITION, "a_position");
    glBindAttribLocation(shaderProgram_sphere, MVD_ATTRIBUTE_NORMAL, "a_normal");

    // Link the program
    glLinkProgram(shaderProgram_sphere);

    status = 0;
    infoLogLength = 0;
    log = NULL;

    glGetProgramiv(shaderProgram_sphere, GL_LINK_STATUS, &status);

    if (status == GL_FALSE)
    {
        glGetProgramiv(shaderProgram_sphere, GL_INFO_LOG_LENGTH, &infoLogLength);

        if (infoLogLength > 0)
        {
            log = (char*) malloc(infoLogLength);

            if (log != NULL)
            {
                GLsizei written;

                glGetProgramInfoLog(shaderProgram_sphere, infoLogLength, &written, log);
                fprintf(gpFile, "Sphere Shader Program Link Log: %s\n", log);
                free(log);
            }
        }
    }
#pragma endregion

    getUniformsForShaderProgramForFragment();
    // Declaration of vertex data arrays
    
    getSphereVertexData(sphere_vertices, sphere_normals, sphere_textures, sphere_elements);
    numVertices_sphere = getNumberOfSphereVertices();
    numElements_sphere = getNumberOfSphereElements();

    recordAndBindBuffers();

    // Required Depth and clear color related changes
    glClearDepth(1.0f);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL);

    // Here Starts OpenGL code
    // this doesn't actually Clear, but actually tells that which Color (blue in  this case) 
    // to be used when we do actual clear
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

    lights[0].lightAmbient = vmath::vec4(0.0f, 0.0f, 0.0f, 1.0f);
    lights[0].lightDiffused = vmath::vec4(1.0f, 0.0f, 0.0f, 1.0f);
    lights[0].lightSpecular = vmath::vec4(1.0f, 0.0f, 0.0f, 1.0f);
    lights[0].lightPosition = vmath::vec4(0.0f, 0.0f, 0.0f, 1.0f);

    lights[1].lightAmbient = vmath::vec4(0.0f, 0.0f, 0.0f, 1.0f);
    lights[1].lightDiffused = vmath::vec4(0.0f, 1.0f, 0.0f, 1.0f);
    lights[1].lightSpecular = vmath::vec4(0.0f, 1.0f, 0.0f, 1.0f);
    lights[1].lightPosition = vmath::vec4(0.0f, 0.0f, 0.0f, 1.0f);

    lights[2].lightAmbient = vmath::vec4(0.0f, 0.0f, 0.0f, 1.0f);
    lights[2].lightDiffused = vmath::vec4(0.0f, 0.0f, 1.0f, 1.0f);
    lights[2].lightSpecular = vmath::vec4(0.0f, 0.0f, 1.0f, 1.0f);
    lights[2].lightPosition = vmath::vec4(0.0f, 0.0f, 0.0f, 1.0f);

    perspectiveProjectionMatrix_sphere = mat4::identity();

    resize_sphere(FBO_WIDTH, FBO_HEIGHT);

    return(0);
}

void recordAndBindBuffers()
{
    // vao_sphere and vbo_position related code
    glGenVertexArrays(1, &vao_sphere);
    glBindVertexArray(vao_sphere);

    glGenBuffers(1, &vbo_position_sphere);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_position_sphere);

    glBufferData(GL_ARRAY_BUFFER, sizeof(sphere_vertices), sphere_vertices, GL_STATIC_DRAW);
    glVertexAttribPointer(MVD_ATTRIBUTE_POSITION, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(MVD_ATTRIBUTE_POSITION);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glGenBuffers(1, &vbo_normal_sphere);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_normal_sphere);

    glBufferData(GL_ARRAY_BUFFER, sizeof(sphere_normals), sphere_normals, GL_STATIC_DRAW);
    glVertexAttribPointer(MVD_ATTRIBUTE_NORMAL, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(MVD_ATTRIBUTE_NORMAL);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // element vbo
    glGenBuffers(1, &vbo_elements_sphere);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vbo_elements_sphere);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(sphere_elements), sphere_elements, GL_STATIC_DRAW);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

    glBindVertexArray(0);
}

void getUniformsForShaderProgramForFragment()
{
    modelMatrixUniform__sphere = glGetUniformLocation(shaderProgram_sphere, "u_modelMatrix");
    viewMatrixUniform__sphere = glGetUniformLocation(shaderProgram_sphere, "u_viewMatrix");
    projectionMatrixUniform__sphere = glGetUniformLocation(shaderProgram_sphere, "u_projectionMatrix");

    laUniform_sphere[0] = glGetUniformLocation(shaderProgram_sphere, "u_la[0]");
    ldUniform_sphere[0] = glGetUniformLocation(shaderProgram_sphere, "u_ld[0]");
    lsUniform_sphere[0] = glGetUniformLocation(shaderProgram_sphere, "u_ls[0]");
    lightPositionUniform_sphere[0] = glGetUniformLocation(shaderProgram_sphere, "u_lightPosition[0]");

    laUniform_sphere[1] = glGetUniformLocation(shaderProgram_sphere, "u_la[1]");
    ldUniform_sphere[1] = glGetUniformLocation(shaderProgram_sphere, "u_ld[1]");
    lsUniform_sphere[1] = glGetUniformLocation(shaderProgram_sphere, "u_ls[1]");
    lightPositionUniform_sphere[1] = glGetUniformLocation(shaderProgram_sphere, "u_lightPosition[1]");

    laUniform_sphere[2] = glGetUniformLocation(shaderProgram_sphere, "u_la[2]");
    ldUniform_sphere[2] = glGetUniformLocation(shaderProgram_sphere, "u_ld[2]");
    lsUniform_sphere[2] = glGetUniformLocation(shaderProgram_sphere, "u_ls[2]");
    lightPositionUniform_sphere[2] = glGetUniformLocation(shaderProgram_sphere, "u_lightPosition[2]");    
    

    kaUniform_sphere = glGetUniformLocation(shaderProgram_sphere, "u_ka");
    kdUniform_sphere = glGetUniformLocation(shaderProgram_sphere, "u_kd");
    ksUniform_sphere = glGetUniformLocation(shaderProgram_sphere, "u_ks");
    materiaShininessUniform_sphere = glGetUniformLocation(shaderProgram_sphere, "u_materialShininess");

    lightingEnabledUniform_sphere = glGetUniformLocation(shaderProgram_sphere, "u_lightingEnabled");
}

void resize(int width, int height)
{
    if(height == 0)
        height = 1;
    winWidth = width;
    winHeight = height;
    // Code
    glViewport(0, 0, (GLsizei) width, (GLsizei)height);
    perspectiveProjectionMatrix = 
    vmath::perspective(45.0f, (GLfloat)width/(GLfloat)height, 0.1f, -100.0f);
}

void resize_sphere(int width, int height)
{
    if(height == 0)
        height = 1;
    // Code
    glViewport(0, 0, (GLsizei) width, (GLsizei)height);
    perspectiveProjectionMatrix_sphere = 
    vmath::perspective(45.0f, (GLfloat)width/(GLfloat)height, 0.1f, -100.0f);
}

void genCPUTexture(void)
{
    glGenTextures(1, &texture_checkerboard);
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1); 
    glBindTexture(GL_TEXTURE_2D, texture_checkerboard);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);
}

void cpuConvolution(float* myArray)
{
     // ================ CPU Convolution Start
    for (int x  = 0; x  < FBO_WIDTH; x ++)
    {
        for (int y  = 0; y  < FBO_HEIGHT; y ++)
        {
            //for (int threadId_y = 0; threadId_y < 16 ; threadId_y++)
			{
			//	for (int threadId_x = 0; threadId_x < 16; threadId_x++)
				{
                    // int tx = threadId_x;
                    // int ty = threadId_y;
                    // int bw = blockId_x;
                    // int bh = blockId_y;
                    // int x = blockId_x * bw + tx;
                    // int y = blockId_y * bh + ty;
                    
                    //cpuConvolutionArray[y * FBO_WIDTH+ x] = myArray [y*FBO_WIDTH+x];

                    float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
                    float rValue = 0, gValue = 0.0f, bValue = 0.0f;
                    int sample = 0;
                    for (int i = -blur_radius; i <= blur_radius; ++i) {
                        for (int j = -blur_radius; j <= blur_radius; ++j) {
                            int c_y = y + i;
                            int c_x = x + j;

                            if (c_x < 0 || c_x >(FBO_WIDTH - 1) || c_y < 0 || (c_y >(FBO_HEIGHT - 1)))
                            {
                                rValue = 0; gValue = 0; bValue = 0;
                            }
                            else
                            {
                                rValue = myArray[(c_y*FBO_WIDTH + c_x) * 4 + 0];
                                gValue = myArray[(c_y*FBO_WIDTH + c_x) * 4 + 1];
                                bValue = myArray[(c_y*FBO_WIDTH + c_x) * 4 + 2];
                            }
                            rSum += rValue * kernel[(i + blur_radius)* 3 + (j + blur_radius)];
                            gSum += gValue * kernel[(i + blur_radius)* 3 + (j + blur_radius)];
                            bSum += bValue * kernel[(i + blur_radius)* 3 + (j + blur_radius)];
                            sample += 1;
                        }
                    }
                    // cpuConvolutionArray[x][y][0] = (GLubyte) remap(myArray [(y*FBO_WIDTH + x) * 4 + 0], 0, 1, 0, 255);
                    // cpuConvolutionArray[x][y][1] = (GLubyte) remap(myArray [(y*FBO_WIDTH + x) * 4 + 1] , 0, 1, 0, 255);
                    // cpuConvolutionArray[x][y][2] = (GLubyte) remap(myArray [(y*FBO_WIDTH + x) * 4 + 2] , 0, 1, 0, 255);
                    // cpuConvolutionArray[x][y][3] = (GLubyte) 255;

                    // cpuConvolutionArray[(y*FBO_WIDTH + x) * 4 + 0] = (GLubyte)remap(rSum / sample, 0, 1, 0, 255);
                    // cpuConvolutionArray[(y*FBO_WIDTH + x) * 4 + 1] = (GLubyte)remap(gSum / sample, 0, 1, 0, 255);
                    // cpuConvolutionArray[(y*FBO_WIDTH + x) * 4 + 2] = (GLubyte)remap(bSum / sample, 0, 1, 0, 255);
                    // cpuConvolutionArray[(y*FBO_WIDTH + x) * 4 + 3] = (GLubyte)255;

                    cpuConvolutionArray[x][y][0] = (GLubyte)remap(rSum / sample, 0, 1, 0, 255);
                    cpuConvolutionArray[x][y][1] = (GLubyte) (GLubyte)remap(gSum / sample, 0, 1, 0, 255);
                    cpuConvolutionArray[x][y][2] = (GLubyte) (GLubyte)remap(bSum / sample, 0, 1, 0, 255);
                    cpuConvolutionArray[x][y][3] = (GLubyte) 255;
                }
            }
        }
        
    }
}

void cpuConvolutionv2(float* myArray)
{
    for (int blockId_x  = 0; blockId_x < FBO_WIDTH/16; blockId_x ++)
    {
        for (int blockId_y  = 0; blockId_y < FBO_HEIGHT/16; blockId_y++)
        {
            for (int threadId_y = 0; threadId_y < 16 ; threadId_y++)
			{
			    for (int threadId_x = 0; threadId_x < 16; threadId_x++)
				{
                    int tx = threadId_x;
                    int ty = threadId_y;
                    int bw = 16;
                    int bh = 16;
                    int x = blockId_x * bw + tx;
                    int y = blockId_y * bh + ty;
                    

                    cpuConvolutionArray[x][y][0] = (GLubyte) remap(myArray [(y*FBO_WIDTH + x) * 4 + 0], 0, 1, 0, 255);
                    cpuConvolutionArray[x][y][1] = (GLubyte) remap(myArray [(y*FBO_WIDTH + x) * 4 + 1] , 0, 1, 0, 255);
                    cpuConvolutionArray[x][y][2] = (GLubyte) remap(myArray [(y*FBO_WIDTH + x) * 4 + 2] , 0, 1, 0, 255);
                    cpuConvolutionArray[x][y][3] = (GLubyte) 255;


                }
            }
        }
        
    }
}
void display(void)
{
    void display_sphere(GLint, GLint);
    void update_sphere(void);
    void processImage(void);
    if (bfboResult)
    {
        display_sphere(FBO_WIDTH, FBO_HEIGHT);
        update_sphere();
        if (enable_cuda_postProcess)
        {
            processImage();    
        }
        else
        {
            
        }
        
    }
    
    glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
    resize(winWidth, winHeight);
    // Code
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // Use the Shader Program Object
    glUseProgram(shaderProgramObj);

    // Triangle

    // Transformations
    mat4 translationMatrix = mat4::identity();
    mat4 rotationMatrix = mat4::identity();
    mat4 modelViewMatrix = mat4::identity();
    mat4 modelViewProjectionMatrix = mat4::identity();

    // Cube    
    // Transformations
    mat4 scaleMatrix = mat4::identity();
    mat4 rotationMatrix_x = mat4::identity();
    mat4 rotationMatrix_y = mat4::identity();
    mat4 rotationMatrix_z = mat4::identity();
    rotationMatrix = mat4::identity();
    modelViewMatrix = mat4::identity();
    modelViewProjectionMatrix = mat4::identity();

    // glTranslatef from FFP is replaced with below line
    translationMatrix = vmath::translate(0.0f, 0.0f, -4.0f);
    scaleMatrix = vmath::scale(0.75f, 0.75f, 0.75f);
    rotationMatrix_x = vmath::rotate(angleCube, 1.0f, 0.0f, 0.0f);
    rotationMatrix_y = vmath::rotate(angleCube, 0.0f, 1.0f, 0.0f);
    rotationMatrix_y = vmath::rotate(angleCube, 0.0f, 0.0f, 1.0f);
    rotationMatrix = rotationMatrix_x * rotationMatrix_y * rotationMatrix_z;
    modelViewMatrix = translationMatrix * scaleMatrix * rotationMatrix;
    modelViewProjectionMatrix = perspectiveProjectionMatrix * modelViewMatrix;

    glUniformMatrix4fv(mvpMatrixUniform, 1, GL_FALSE, modelViewProjectionMatrix);
    glActiveTexture(GL_TEXTURE0);
    if (enable_cuda_postProcess)
    {
        glBindTexture(GL_TEXTURE_2D, tex_cudaResult);
    }
    else
    {
        //glBindTexture(GL_TEXTURE_2D, fbo_texture);
        float *new_array = (float *)malloc(FBO_WIDTH * FBO_HEIGHT * 4 * sizeof(float));
            glActiveTexture(GL_TEXTURE0);
            glBindTexture(GL_TEXTURE_2D, fbo_texture);
            /* get texture data from video memory */
            glGetTexImage(GL_TEXTURE_2D, 0, GL_RGBA, GL_FLOAT, (void*)(new_array));
            glBindTexture(GL_TEXTURE_2D, 0);

                      
            glActiveTexture(GL_TEXTURE0);
            glBindTexture(GL_TEXTURE_2D, texture_checkerboard);            
            
            // Below function is deprecated
            //glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

            //glBindTexture(GL_TEXTURE_2D, 0);

           // CPU Convoultion call should be here
           cpuConvolutionv2(new_array);
           glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, FBO_WIDTH, FBO_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void *)cpuConvolutionArray);
    }
    //
    glUniform1i(textureSamplerUniform, 0);
    glBindVertexArray(vao_cube);
    
    // Here there should be the drawing of Graphics / Scenes / Animation
    glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
    glDrawArrays(GL_TRIANGLE_FAN, 4, 4);
    glDrawArrays(GL_TRIANGLE_FAN, 8, 4);
    glDrawArrays(GL_TRIANGLE_FAN, 12, 4);
    glDrawArrays(GL_TRIANGLE_FAN, 16, 4);
    glDrawArrays(GL_TRIANGLE_FAN, 20, 4);
    glBindVertexArray(0);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Un-use the Program
    glUseProgram(0);
    hipDeviceSynchronize();
    SwapBuffers(ghdc);
}

void update(void)
{
    // Code
    
    angleCube += 1.0f;
    if (angleCube >= 360.0f)
    {
        angleCube -= 360.0f;
    }
}

void display_sphere(GLint textureWidth, GLint textureHeight)
{
    // Code
    glBindFramebuffer(GL_FRAMEBUFFER, fbo);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    resize_sphere(textureWidth, textureHeight);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // Use the Shader Program Object
    glUseProgram(shaderProgram_sphere);
    
    // Transformations
    mat4 translationMatrix = mat4::identity();
    //mat4 modelViewMatrix = mat4::identity();
    mat4 modelMatrix = mat4::identity();
    mat4 viewMatrix = mat4::identity();

    // glTranslatef from FFP is replaced with below line
    translationMatrix = vmath::translate(0.0f, 0.0f, -2.0f);
    modelMatrix = translationMatrix;

    glUniformMatrix4fv(modelMatrixUniform__sphere, 1, GL_FALSE, modelMatrix);
    glUniformMatrix4fv(viewMatrixUniform__sphere, 1, GL_FALSE, viewMatrix);
    glUniformMatrix4fv(projectionMatrixUniform__sphere, 1, GL_FALSE, perspectiveProjectionMatrix_sphere);

    // Sending light related uniforms
    if (bLight == TRUE)
    {
        lights[0].lightPosition[1] = 15 * -sinf(lightAngleZero_sphere);
        lights[0].lightPosition[2] = 15 * cos(-lightAngleZero_sphere);

        lights[1].lightPosition[0] =  15 * cosf(-lightAngleOne_sphere);
        lights[1].lightPosition[2] =  15 * -sinf(lightAngleOne_sphere);

        lights[2].lightPosition[0] =  15 * -sin(-lightAngleTwo_sphere);
        lights[2].lightPosition[1] =  15 * -cos(lightAngleTwo_sphere);
        
        {
            glUniform1i(lightingEnabledUniform_sphere, 1);
            for (int i = 0; i < 3; i++)
            {
                glUniform3fv(laUniform_sphere[i], 1, lights[i].lightAmbient);
                glUniform3fv(ldUniform_sphere[i], 1, lights[i].lightDiffused);
                glUniform3fv(lsUniform_sphere[i], 1, lights[i].lightSpecular);
                glUniform4fv(lightPositionUniform_sphere[i], 1, lights[i].lightPosition);
            }

            glUniform3fv(kaUniform_sphere, 1, materialAmbient_sphere);
            glUniform3fv(kdUniform_sphere, 1, materialDiffused_sphere);
            glUniform3fv(ksUniform_sphere, 1, materialSpecular_sphere);
            glUniform1f(materiaShininessUniform_sphere, materialShininess_sphere);

            glUniformMatrix4fv(modelMatrixUniform__sphere, 1, GL_FALSE, modelMatrix);
            glUniformMatrix4fv(viewMatrixUniform__sphere, 1, GL_FALSE, viewMatrix);
            glUniformMatrix4fv(projectionMatrixUniform__sphere, 1, GL_FALSE, perspectiveProjectionMatrix_sphere);
        }        
    }
    else
    {
        glUniform1i(lightingEnabledUniform_sphere, 0);
        glUniform1i(lightingEnabledUniform_sphere, 0);
    }

    glBindVertexArray(vao_sphere);

    // *** draw, either by glDrawTriangles() or glDrawArrays() or glDrawElements()
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vbo_elements_sphere);
    glDrawElements(GL_TRIANGLES, numElements_sphere, GL_UNSIGNED_SHORT, 0);

    glBindVertexArray(0);

    // Un-use the Program
    glUseProgram(0);
    glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

void update_sphere(void)
{
    // Code
    lightAngleZero_sphere += 0.05f;
    if (lightAngleZero_sphere >= 360.0f)
    {
        lightAngleZero_sphere -= 360.0f;
    }
    
    lightAngleOne_sphere += 0.05f;
    if (lightAngleOne_sphere >= 360.0f)
    {
        lightAngleOne_sphere -= 360.0f;
    }

    lightAngleTwo_sphere += 0.05f;
    if (lightAngleTwo_sphere >= 360.0f)
    {
        lightAngleTwo_sphere -= 360.0f;
    }
}

void uninitialize(void)
{
    void uninitialize_sphere(void);
    GLsizei numAttachedShaders;
    // Function declarations
    void ToggleFullScreen(void);

    //Code
    // Convention not compulsion
    // Because user can press escape in Full screen mode and OS takes lot of pain to make it full screen
    // Yasya Gruhe Mata Nasti, Tasy gruhe haritaki (Hirada)
    if(gbFullScreen)
    {
        ToggleFullScreen();
    }
    uninitialize_sphere();
    // Deletion and uninitialization of vbo_position
     if (fbo)
    {
        glDeleteFramebuffers(1, &fbo);
        fbo = 0;
    }
     if (rbo)
    {
        glDeleteRenderbuffers(1, &rbo);
        rbo = 0;
    }
    if (fbo_texture)
    {
        glDeleteTextures(1, &fbo_texture);
        fbo_texture = 0;
    }
    if (vbo_cube_texcoord)
    {
        glDeleteBuffers(1, &vbo_cube_texcoord);
        vbo_cube_texcoord = 0;
    }
    if (vbo_cube_position)
    {
        glDeleteBuffers(1, &vbo_cube_position);
        vbo_cube_position = 0;
    }
    // Deletion and uninitialization of vao
    if (vao_cube)
    {
        glDeleteVertexArrays(1, &vao_cube);
        vao_cube = 0;
    }
    // Shader Uninitalization
    if (shaderProgramObj)
    {
        // Use program
        glUseProgram(shaderProgramObj);

        // Get the number of Attached shaders
        glGetProgramiv(shaderProgramObj, GL_ATTACHED_SHADERS, &numAttachedShaders);
        
        GLuint *shaderObjects = NULL;
        shaderObjects = (GLuint*) malloc(sizeof(GLuint) * numAttachedShaders);
        
        // Fill empty buffer with attached shared the objects
        glGetAttachedShaders(shaderProgramObj, numAttachedShaders, &numAttachedShaders, shaderObjects);
        
        // Loop the attached shaders, detach each shader and then delete each shader
        for (GLsizei i = 0; i < numAttachedShaders; i++)
        {
            glDetachShader(shaderProgramObj, shaderObjects[i]);
            glDeleteShader(shaderObjects[i]);
            shaderObjects[i] = 0;
        }
        
        free(shaderObjects);
        shaderObjects = NULL;

        // Un-use the program
        glUseProgram(0);

        // Delete the Program object
        glDeleteProgram(shaderProgramObj);
        shaderProgramObj = 0;
    }
    
    if (wglGetCurrentContext() == ghrc)
    {
        // Get the responsibilities out from ghrc
        wglMakeCurrent(NULL, NULL);
    }

    if (ghrc)
    {
        // Delete the the ghrc
        wglDeleteContext(ghrc);
        ghrc = NULL;
    }

    if (ghdc)
    {
        ReleaseDC(ghwnd, ghdc);
        ghdc = NULL;
    }
    
    if(ghwnd)
    {
        DestroyWindow(ghwnd);
        ghwnd = NULL;
    }
    
    if (gpFile)
    {
        fprintf(gpFile, "Log File Is Closed Successfully.\n");
        fclose(gpFile);
        gpFile = NULL;
    }
}

void uninitialize_sphere(void)
{
    void deleteProgram(GLuint, GLsizei);
    GLsizei numAttachedShaders;
    
    // Deletion and uninitialization of vbo_position
    if (vbo_elements_sphere)
    {
        glDeleteBuffers(1, &vbo_elements_sphere);
        vbo_elements_sphere = 0;
    }
    if (vbo_normal_sphere)
    {
        glDeleteBuffers(1, &vbo_normal_sphere);
        vbo_normal_sphere = 0;
    }
    if (vbo_position_sphere)
    {
        glDeleteBuffers(1, &vbo_position_sphere);
        vbo_position_sphere = 0;
    }
    
    // Deletion and uninitialization of vao_sphere
    if (vao_sphere)
    {
        glDeleteVertexArrays(1, &vao_sphere);
        vao_sphere = 0;
    }
    // Shader Uninitalization    
    if (shaderProgram_sphere)
    {
       deleteProgram(shaderProgram_sphere, numAttachedShaders);
    }    
}

void deleteProgram(GLuint shaderProgramObj, GLsizei numAttachedShaders)
{
    // Use program
        glUseProgram(shaderProgramObj);

        // Get the number of Attached shaders
        glGetProgramiv(shaderProgramObj, GL_ATTACHED_SHADERS, &numAttachedShaders);
        
        GLuint *shaderObjects = NULL;
        shaderObjects = (GLuint*) malloc(sizeof(GLuint) * numAttachedShaders);
        
        // Fill empty buffer with attached shared the objects
        glGetAttachedShaders(shaderProgramObj, numAttachedShaders, &numAttachedShaders, shaderObjects);
        
        // Loop the attached shaders, detach each shader and then delete each shader
        for (GLsizei i = 0; i < numAttachedShaders; i++)
        {
            glDetachShader(shaderProgramObj, shaderObjects[i]);
            glDeleteShader(shaderObjects[i]);
            shaderObjects[i] = 0;
        }
        
        free(shaderObjects);
        shaderObjects = NULL;

        // Un-use the program
        glUseProgram(0);

        // Delete the Program object
        glDeleteProgram(shaderProgramObj);
}
